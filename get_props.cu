
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{

	int device;
	hipGetDevice(&device);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	// just a few random properties 
	printf("Here is a few random cuda device properties ... \n");
	printf("Name: %s\n", deviceProp.name);
	printf("Regs per block: %d\n", deviceProp.regsPerBlock);
	printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Max grid size: %d\n", deviceProp.maxGridSize[0]);
	printf("Multi-proc count: %d\n", deviceProp.multiProcessorCount);
	printf("ECC enabled: %d\n", deviceProp.ECCEnabled);
	printf("Memory bus witdh: %d\n", deviceProp.memoryBusWidth);
	printf("Max threads per multi-proc: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("Warp size: %d \n", deviceProp.warpSize);

	hipDeviceReset();

	return 0;
}