
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define BLOCK_DIM 32

__global__ void kernel(int *vec_a, int *vec_b, int vec_dim)
{

	__shared__ int sh_vec_a[BLOCK_DIM + 2];

	int grid_dim = gridDim.x * blockDim.x;

	// some element calculation based on the few elements around

	int step = 0;
	int steps_cnt = (vec_dim + (2 * gridDim.x) + grid_dim) / grid_dim;
	while (step < steps_cnt)
	{
		int gl_ind = (step * grid_dim) + blockDim.x * blockIdx.x + threadIdx.x;
		int stride_ind = gl_ind - blockIdx.x * 2 - step * gridDim.x * 2;

		if (stride_ind < vec_dim)
		{
			sh_vec_a[threadIdx.x] = vec_a[stride_ind];
		}
		__syncthreads();

		if (stride_ind < vec_dim)
		{
			if (threadIdx.x < BLOCK_DIM - 2)
			{
				int value = sh_vec_a[threadIdx.x] + sh_vec_a[threadIdx.x + 1] + sh_vec_a[threadIdx.x + 2];

				vec_b[stride_ind] = value;
			}
		}

		__syncthreads();

		step++;
	}

	return;
}

void init_vec(int *vec, int len, int value)
{
	for (int i = 0; i < len; i++)
	{
		vec[i] = value;
	}
}

void print_vec(int *vec, int len)
{
	for (int i = 0; i < len; i++)
	{
		printf("%d ,", vec[i]);
	}
	printf("\n");
}

int main(void)
{

	int vec_dim = 1000;

	int *vec_a = (int *)malloc(vec_dim * sizeof(int));
	int *vec_b = (int *)malloc(vec_dim * sizeof(int));

	init_vec(vec_a, vec_dim, 1);
	init_vec(vec_b, vec_dim, 0);

	int *dev_vec_a;
	int *dev_vec_b;
	hipMalloc((void **)&dev_vec_a, vec_dim * sizeof(vec_dim));
	hipMalloc((void **)&dev_vec_b, vec_dim * sizeof(vec_dim));

	hipMemcpy(dev_vec_a, vec_a, vec_dim * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_vec_b, vec_b, vec_dim * sizeof(int), hipMemcpyHostToDevice);

	// this will ensure that at least one block is spawned if vec_dim < BLOCK_DIM
	int grid_dim = (vec_dim + BLOCK_DIM) / BLOCK_DIM;

	kernel<<<1, BLOCK_DIM>>>(dev_vec_a, dev_vec_b, vec_dim);

	hipMemcpy(vec_a, dev_vec_a, vec_dim * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(vec_b, dev_vec_b, vec_dim * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_vec_a);
	hipFree(dev_vec_b);

	printf("Just the samples (10) ... \n");
	print_vec(vec_a, 10);
	print_vec(vec_b, 10);

	free(vec_a);
	free(vec_b);

	return 0;
}
